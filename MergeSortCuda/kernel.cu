#include "hip/hip_runtime.h"
//cuda libs

#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "thrust\merge.h"
//local lib
#include "Helper.h"
#include "MergeSort.h"
#include "MergeSortMultiThreading.h"

//============================MERGE SORT FREE THREADS========================
//Number of threads per block
#define THREADS_PER_BLOCK 512
//compare 2 number, return the minimum
__device__ long min_int(long, long);
//Bottom up merging implementation on gpu device
//Merge 2 parts of array, [l, m) and [m, r)
__device__ void free_merge_bottom_up(int*, int*, long, long, long);
//This function is called by every thread
//each thread will calculate its range from its id, and starting merge on that range
__global__ void gpu_free_thread_merge_sort(int*, int*, long, long);
//Mail call
double gpuMergeSortFreeThreads(int*, long);
//============================================================================
//=============================MERGE SORT FIXED THREADS=======================
//Max thread on single block
#define MAX_THREADS 8
//Copy a part of array B to A
__device__ void fixed_CopyArray(int*, int*, long, long);
//Merge 2 parts of array A (left and right), result save to array B. Split by mid.
__device__ void fixed_BottomUpMerge(int*, int*, long, long , long );
//Array A[] has the items to sort; array B[] is a temporary array
//This function can sort a part of array, start from left and end at right.
__device__ void fixed_BottomUpMergeSort(int*, int* , long, long );
//Thread sort on its part
__global__ void fixed_thread_mergeSort(int* , int* , long );
//Thread merge
__global__ void fixed_thread_merge(int* , int* , long* , long* , long );
//Main call
double gpuMergeSortFixedThreads(int*, long);
//============================MERGESORT USING CUDA TOOLKIT=====================
void thrust_merge(int*a, int left, int mid, int right);
double thrust_BottomUpMergeSort(int* A, long n);
//============================MERGE SORT ON CPU================================
//Merge sort normal
double mergeSortNormal(int*, long);
//Merge sort multi-thread
double mergeSortMT(int*, long);
//-----------------------------------------------------------------------------
//every test case, do 5 times and get average time
void analysis(long length) {

	int n_times = 5;

	double avg_time_1 = 0;
	double avg_time_2 = 0;
	double avg_time_3 = 0;
	double avg_time_4 = 0;
	double avg_time_5 = 0;

	for (int i = 0; i < n_times; i++) {
		//create sample
		int* sample;
		bool g = generateRandomList(length, sample);
		if (!g)
		{
			cout << "Cannot create sample array!\n";
			return;
		}
		//1 thread merge sort	
		avg_time_1 += mergeSortNormal(sample, length);
		//9 threads merge sort
		avg_time_2 += mergeSortMT(sample, length);
		//cuda free threads
		avg_time_3 += gpuMergeSortFreeThreads(sample, length);
		//cuda fixed threads
		avg_time_4 += gpuMergeSortFixedThreads(sample, length);
		//cuda toolkit merge
		avg_time_5 += thrust_BottomUpMergeSort(sample, length);
		//
		free(sample);
	}
	avg_time_1 /= n_times;
	avg_time_2 /= n_times;
	avg_time_3 /= n_times;
	avg_time_4 /= n_times;
	avg_time_5 /= n_times;
	//write log
	cout.precision(dbl::max_digits10);
	cout << length << "\t" << fixed << avg_time_1 << "\t" << fixed << avg_time_2 << "\t" << fixed << avg_time_3 << "\t" << fixed << avg_time_4 << "\t" << fixed << avg_time_5 << endl;
	writeLog("result.txt", length, avg_time_1, avg_time_2, avg_time_3, avg_time_4, avg_time_5);
}

void  main() {
	srand(time(NULL));
	//Sample size
	long sample_size = 100;
	//first test case
	analysis(sample_size);
	//19 remaining test case
	for (int i = 1; i < 20; i++) {
		sample_size += (sample_size / 2);//new test case input is larger than old one 50%
		analysis(sample_size);
	}
	cout << endl;
	system("pause");
}
//=============================IMPLEMENTATION=======================
__device__ long min_int(long a, long b) {
	return (a < b) ? a : b;
}

//Merge sort normal
double mergeSortNormal(int* a, long length) {
	//create temp list
	int *temp = (int*)malloc(sizeof(int) * length);
	//
	high_resolution_clock::time_point watch = high_resolution_clock::now();
	BottomUpMergeSort(a, temp, 0, length);
	duration<double> time_span = (high_resolution_clock::now() - watch);
	//free temp
	free(temp);
	//return execution time
	return time_span.count();
}

//Merge sort multi-thread
double mergeSortMT(int* a, long length) {
	//create temp list
	int *temp = (int*)malloc(sizeof(int) * length);
	//
	high_resolution_clock::time_point watch = high_resolution_clock::now();
	MultiThreadingMergeSort(a, temp, length);
	duration<double> time_span = (high_resolution_clock::now() - watch);
	//free temp
	free(temp);
	//return execution time
	return time_span.count();
}

//====================================CUDA TOOLKIT MERGE SORT==========================
//Merge 2 parts of array using cuda toolkit
void thrust_merge(int*a, int left, int mid, int right) {
	int* temp = new int[right - left];
	thrust::merge(&a[left], &a[left] + (mid - left), &a[mid], &a[mid] + (right - mid), temp);

	//copy to main array
	for (int i = 0; i < right - left; i++)
		a[left + i] = temp[i];

	delete[] temp;
}

//Bottom up merge sort using cuda toolkit
double thrust_BottomUpMergeSort(int* A, long n)
{
	high_resolution_clock::time_point watch = high_resolution_clock::now();
	//
	for (int width = 1; width < n; width = 2 * width)
		for (int i = 0; i < n; i = i + 2 * width)
			thrust_merge(A, i, min(i + width, n), min(i + 2 * width, n));
	//
	duration<double> time_span = (high_resolution_clock::now() - watch);
	return time_span.count();
}

//Copy a part of array B to A
__device__ void fixed_CopyArray(int* B, int* A, long l, long r)
{
	for (int i = l; i < r; i++)
		A[i] = B[i];
}

//Merge 2 parts of array A (left and right), result save to array B. Split by mid.
__device__ void fixed_BottomUpMerge(int* A, int* B, long left, long mid, long right)
{
	int i = left; int j = mid;
	// While there are elements in the left or right runs...
	for (int k = left; k < right; k++) {
		// If left run head exists and is <= existing right run head.
		if (i < mid && (j >= right || A[i] <= A[j])) {
			B[k] = A[i];
			i = i + 1;
		}
		else {
			B[k] = A[j];
			j = j + 1;
		}
	}
}

__device__ void fixed_BottomUpMergeSort(int* A, int* B, long l, long r)
{
	int n = r - l;
	// Each 1-element run in A is already "sorted".
	// Make successively longer sorted runs of length 2, 4, 8, 16... until whole array is sorted.
	for (long width = 1; width < n; width = 2 * width)
	{
		// Array A is full of runs of length width.
		for (long i = l; i < r; i = i + 2 * width)
		{
			// Merge two runs: A[i:i+width-1] and A[i+width:i+2*width-1] to B[]
			// or copy A[i:n-1] to B[] ( if(i+width >= n) )
			fixed_BottomUpMerge(A, B, i, min_int(i + width, r), min_int(i + 2 * width, r));
		}
		// Now work array B is full of runs of length 2*width.
		// Copy array B to array A for next iteration.
		// A more efficient implementation would swap the roles of A and B.
		fixed_CopyArray(B, A, l, r);
		//SwapArray(A, B);
		// Now array A is full of runs of length 2*width.
	}
}

//Thread sort on its part
__global__ void fixed_thread_mergeSort(int* a, int* temp, long n) {
	//Get current ThreadID
	int ThreadID = threadIdx.x;
	//Get the job which is given to thread
	long from = ThreadID *  (n / MAX_THREADS);
	//Sort on that range
	fixed_BottomUpMergeSort(a, temp, from,
		from + ((ThreadID == MAX_THREADS - 1) ? (n / MAX_THREADS) + (n % MAX_THREADS) : (n / MAX_THREADS)));
}

//Thread merge
__global__ void fixed_thread_merge(int* a, int* temp, long* from, long* to, long loc_size) {
	int id = threadIdx.x * 2;

	fixed_BottomUpMerge(a, temp, from[id], to[id], to[id + 1]);
	fixed_CopyArray(temp, a, from[id], to[id + 1]);
	long cur = from[id];
	long cur1 = to[id + 1];
	//
	from[threadIdx.x] = cur;
	to[threadIdx.x] = cur1;
}

double gpuMergeSortFixedThreads(int* a, long n) {
	//device array pointers
	int *dev_working;
	int *dev_temp;
	double exetime = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipSetDevice(0);
	//Allocation gpu memory
	hipMalloc((void**)&dev_temp, sizeof(long) * n);
	hipMalloc((void**)&dev_working, sizeof(long) * n);
	//Copy local array to gpu-memory
	hipMemcpy(dev_working, a, sizeof(long) * n, hipMemcpyHostToDevice);

	high_resolution_clock::time_point watch = high_resolution_clock::now();
	//threads sort on its range
	fixed_thread_mergeSort << <1, MAX_THREADS >> > (dev_working, dev_temp, n);	
	hipDeviceSynchronize();
	//
	duration<double> time_span = (high_resolution_clock::now() - watch);
	exetime += time_span.count();

	//threads job simulation
	long* from = (long*)malloc(sizeof(long) * MAX_THREADS);
	long* to = (long*)malloc(sizeof(long) * MAX_THREADS);
	//
	long n_l_jobs = n / MAX_THREADS;
	long n_l_j_remain = n % MAX_THREADS;
	//calculate job and send to gpu, to merge result above
	for (int i = 0; i < MAX_THREADS; i++) {
		long current_jobs = n_l_jobs;
		if (i == MAX_THREADS - 1) //if it's last thread, give it remain job
			current_jobs += n_l_j_remain;

		from[i] = i * n_l_jobs;
		to[i] = i * n_l_jobs + current_jobs;
	}
	//Copy job detail to gpu
	long *dev_from;
	long *dev_to;
	hipMalloc((void**)&dev_from, sizeof(long) * MAX_THREADS);
	hipMalloc((void**)&dev_to, sizeof(long) * MAX_THREADS);
	hipMemcpy(dev_from, from, sizeof(long) * MAX_THREADS, hipMemcpyHostToDevice);
	hipMemcpy(dev_to, to, sizeof(long) * MAX_THREADS, hipMemcpyHostToDevice);

	watch = high_resolution_clock::now();
	//merge results
	for (int range = 1; range < MAX_THREADS; range *= 2)
		fixed_thread_merge <<<1, MAX_THREADS / range >> > (dev_working, dev_temp, dev_from, dev_to, MAX_THREADS);
	//
	hipDeviceSynchronize();
	//
	time_span = (high_resolution_clock::now() - watch);
	exetime += time_span.count();
	//Copy result from device
	cudaStatus = hipMemcpy(a, dev_working, sizeof(long) * n, hipMemcpyDeviceToHost);
	//
	hipFree(dev_working);	hipFree(dev_temp); hipFree(dev_from);	hipFree(dev_to);
	//return execution time
	return time_span.count();
}

//==================================MERGE SORT FREE THREADS=======================
//Bottom up merging implementation on gpu device
//Merge 2 parts of array, [l, m) and [m, r)
__device__ void free_merge_bottom_up(int* a, int* temp, long l, long m, long r) {
	int i = l, j = m;
	// While there are elements in the left or right runs...
	for (int k = l; k < r; k++) {
		// If left run head exists and is <= existing right run head.
		if (i < m && (j >= r || a[i] <= a[j])) {
			temp[k] = a[i];
			i = i + 1;
		}
		else {
			temp[k] = a[j];
			j = j + 1;
		}
	}
}
//This function is called by every thread
//each thread will calculate its range from its id, and starting merge on that range
__global__ void gpu_free_thread_merge_sort(int* a, int* temp, long n, long width)
{
	//id of thread
	long idx = blockIdx.x * blockDim.x + threadIdx.x;
	//define working range of thread
	//range begining
	long i = idx * 2 * width;
	if (i >= n)
		return;
	//range detail
	long left = i;
	long mid = min_int(n, i + width);
	long right = min_int(n, i + 2 * width);
	//Do bottom up merge
	free_merge_bottom_up(a, temp, left, mid, right);
}

//This function prepair gpu memory and call kernel function
double gpuMergeSortFreeThreads(int *a, long n) {
	hipSetDevice(0);
	//device array pointers
	int *dev_working;
	int *dev_temp;

	//Allocation gpu memory
	hipMalloc((void**)&dev_temp, sizeof(int) * n);
	hipMalloc((void**)&dev_working, sizeof(int) * n);

	//Copy local array to gpu-memory
	//This line waste a lot of time,
	//algorithm do very quickly but memory copy host-to-device, device-to-host lightly slow.
	hipMemcpy(dev_working, a, sizeof(long) * n, hipMemcpyHostToDevice);

	//
	int width;

	//Temporary array
	int* A = dev_working;
	int* B = dev_temp;

	//Clock for watching time

	high_resolution_clock::time_point watch = high_resolution_clock::now();
	hipError_t err;
	//Split array to ranges, each range has length equal to width
	//width is multiplied by 2
	for (width = 1; width < n; width *= 2) {
		//number of threads need to use
		long n_threads_need = n / width;
		//number of blocks from n_threads_need
		long n_blocks = (n_threads_need + (THREADS_PER_BLOCK - 1)) / (THREADS_PER_BLOCK);
		//call kernel
		gpu_free_thread_merge_sort << <n_blocks, THREADS_PER_BLOCK >> >(A, B, n, width);
		hipDeviceSynchronize();

		//swap array
		A = A == dev_working ? dev_temp : dev_working;
		B = B == dev_working ? dev_temp : dev_working;
	}
	//stop clock
	duration<double> time_span = (high_resolution_clock::now() - watch);

	//Copy result to local memory
	//This line waste alot of time,
	//algorithm is very quick but memory copy host-to-device, device-to-host lightly slow.
	hipMemcpy(a, A, sizeof(long) * n, hipMemcpyDeviceToHost);
	//Free gpu memory
	hipFree(dev_temp);
	hipFree(dev_working);
	//Return execution time of merge sort
	//
	return time_span.count();
}